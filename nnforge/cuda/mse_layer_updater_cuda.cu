#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2015 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "mse_layer_updater_cuda.h"

#include <hip/hip_runtime.h>

#include "util_cuda.h"
#include "../mse_layer.h"
#include "../neural_network_exception.h"

namespace nnforge
{
	namespace cuda
	{
		extern __shared__ float arr_sh[];
		__global__ void mse_upd_kernel(
			float * __restrict output,
			const float * __restrict input0,
			const float * __restrict input1,
			const float * __restrict scale_mask,
			int input_feature_map_count,
			int elem_count_per_feature_map,
			float scale,
			int entry_count)
		{
			int feature_map_id = threadIdx.x;
			int neuron_id = blockIdx.x;
			int entry_id = blockIdx.y;
			int threadblock_size = blockDim.x;

			float err = 0.0F;

			int output_offset = entry_id * elem_count_per_feature_map + neuron_id;

			float mask = 1.0F;
			if (scale_mask)
				mask = scale_mask[output_offset];

			int thread_id = threadIdx.x;
			if (mask != 0.0F)
			{
				int input_offset = (entry_id * input_feature_map_count + feature_map_id) * elem_count_per_feature_map + neuron_id;
				while (feature_map_id < input_feature_map_count)
				{
					float local_err = input0[input_offset] - input1[input_offset];
					err += local_err * local_err;
					feature_map_id += threadblock_size;
					input_offset += threadblock_size * elem_count_per_feature_map;
				}

				int lane_id = thread_id & 31;
				#pragma unroll
				for(int tx = 16; tx > 0; tx >>= 1)
					err += __shfl_down(err, tx);

				int warp_count = threadblock_size >> 5;
				if (warp_count > 1)
				{
					if (lane_id == 0)
						arr_sh[thread_id >> 5] = err;

					__syncthreads();

					if (thread_id < 32)
					{
						err = 0.0F;
						if (thread_id < warp_count)
							err = arr_sh[thread_id];
						#pragma unroll
						for(int tx = 4; tx > 0; tx >>= 1)
							err += __shfl_down(err, tx);
					}
				}
			}
		
			if (thread_id == 0)
				output[output_offset] = err * (mask * scale);
		}

		template<bool add_update_to_destination>
		__global__ void mse_backprop_upd_kernel(
			float * __restrict output,
			const float * __restrict deriv_input_neurons,
			const float * __restrict target_input_neurons,
			float scale2,
			int elem_count) 
		{
			int elem_id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
			if (elem_id < elem_count)
			{
				if (add_update_to_destination)
					output[elem_id] += scale2 * (target_input_neurons[elem_id] - deriv_input_neurons[elem_id]);
				else
					output[elem_id] = scale2 * (target_input_neurons[elem_id] - deriv_input_neurons[elem_id]);
			}
		}

		template<bool add_update_to_destination>
		__global__ void mse_backprop_upd_kernel(
			float * __restrict output,
			const float * __restrict deriv_input_neurons,
			const float * __restrict target_input_neurons,
			const float * __restrict scale_mask,
			float scale2,
			int elem_count_per_feature_map,
			int input_feature_map_count,
			int entry_count) 
		{
			int neuron_id = blockDim.x * blockIdx.x + threadIdx.x;
			int feature_map_id = blockDim.y * blockIdx.y + threadIdx.y;
			int entry_id = blockDim.z * blockIdx.z + threadIdx.z;
			if ((neuron_id < elem_count_per_feature_map) && (feature_map_id < input_feature_map_count) && (entry_id < entry_count))
			{
				int elem_id = (entry_id * input_feature_map_count + feature_map_id) * elem_count_per_feature_map + neuron_id;
				float mask = scale_mask[entry_id * elem_count_per_feature_map + neuron_id];
				float gradient = 0.0F;
				if (mask != 0.0F)
				{
					float actual_val = target_input_neurons[elem_id];
					float predicted_val = deriv_input_neurons[elem_id];
					gradient = actual_val - predicted_val;
				}

				if (add_update_to_destination)
					output[elem_id] += gradient * (mask * scale2);
				else
					output[elem_id] = gradient * (mask * scale2);
			}
		}

		mse_layer_updater_cuda::mse_layer_updater_cuda()
		{
		}

		mse_layer_updater_cuda::~mse_layer_updater_cuda()
		{
		}

		void mse_layer_updater_cuda::enqueue_forward_propagation(
			hipStream_t stream_id,
			cuda_linear_buffer_device::ptr output_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device::ptr>& data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data_custom,
			const std::vector<cuda_linear_buffer_device::const_ptr>& input_buffers,
			const std::vector<cuda_linear_buffer_device::const_ptr>& persistent_working_data,
			cuda_linear_buffer_device::ptr temporary_working_fixed_buffer,
			cuda_linear_buffer_device::ptr temporary_working_per_entry_buffer,
			cuda_linear_buffer_device::ptr temporary_per_entry_buffer,
			unsigned int entry_count)
		{
			int threadblock_size = get_threadblock_size(input_configuration_specific_list[0].feature_map_count);
			const float * scale_mask = 0;
			if (input_buffers.size() > 2)
				scale_mask = *input_buffers[2];

			int smem_size = ((threadblock_size + 32 - 1) / 32) * sizeof(float);
			mse_upd_kernel<<<dim3(input_elem_count_per_feature_map_list[0], entry_count), threadblock_size, smem_size, stream_id>>>(
				*output_buffer,
				*input_buffers[0],
				*input_buffers[1],
				scale_mask,
				input_configuration_specific_list[0].feature_map_count,
				input_elem_count_per_feature_map_list[0],
				scale,
				entry_count);
		}

		void mse_layer_updater_cuda::enqueue_backward_data_propagation(
			hipStream_t stream_id,
			unsigned int input_index,
			cuda_linear_buffer_device::ptr input_errors_buffer,
			cuda_linear_buffer_device::const_ptr output_errors_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device::ptr>& data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data_custom,
			const std::vector<cuda_linear_buffer_device::const_ptr>& input_neurons_buffers,
			cuda_linear_buffer_device::const_ptr output_neurons_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& persistent_working_data,
			cuda_linear_buffer_device::ptr temporary_working_fixed_buffer,
			cuda_linear_buffer_device::ptr temporary_working_per_entry_buffer,
			cuda_linear_buffer_device::const_ptr temporary_per_entry_buffer,
			bool add_update_to_destination,
			unsigned int entry_count)
		{
			if (input_neurons_buffers.size() > 2)
			{
				std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
					*cuda_config,
					input_elem_count_per_feature_map_list[0],
					input_configuration_specific_list[0].feature_map_count,
					entry_count);

				if (add_update_to_destination)
					mse_backprop_upd_kernel<true><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
						*input_errors_buffer,
						*input_neurons_buffers[input_index],
						*input_neurons_buffers[1 - input_index],
						*input_neurons_buffers[2],
						scale * 2.0F,
						input_elem_count_per_feature_map_list[0],
						input_configuration_specific_list[0].feature_map_count,
						entry_count);
				else
					mse_backprop_upd_kernel<false><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
						*input_errors_buffer,
						*input_neurons_buffers[input_index],
						*input_neurons_buffers[1 - input_index],
						*input_neurons_buffers[2],
						scale * 2.0F,
						input_elem_count_per_feature_map_list[0],
						input_configuration_specific_list[0].feature_map_count,
						entry_count);
			}
			else
			{
				int elem_count = entry_count * input_elem_count_per_entry_list[0];
				std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
					*cuda_config,
					elem_count);

				if (add_update_to_destination)
					mse_backprop_upd_kernel<true><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
						*input_errors_buffer,
						*input_neurons_buffers[input_index],
						*input_neurons_buffers[1 - input_index],
						scale * 2.0F,
						elem_count);
				else
					mse_backprop_upd_kernel<false><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
						*input_errors_buffer,
						*input_neurons_buffers[input_index],
						*input_neurons_buffers[1 - input_index],
						scale * 2.0F,
						elem_count);
			}
		}

		void mse_layer_updater_cuda::updater_configured()
		{
			if (actions.find(layer_action(layer_action::backward_data, 2)) != actions.end())
				throw neural_network_exception("mse_layer_updater_cuda cannot do backward propagation for scale mask");

			nnforge_shared_ptr<const mse_layer> layer_derived = nnforge_dynamic_pointer_cast<const mse_layer>(layer_schema);

			scale = layer_derived->scale;
		}

		bool mse_layer_updater_cuda::is_backward_data_dependent_on_output_buffer(unsigned int action_input_index) const
		{
			return false;
		}

		int mse_layer_updater_cuda::get_threadblock_size(int input_feature_map_count)
		{
			int threadblock_size;

			if (input_feature_map_count < 256)
			{
				threadblock_size = (input_feature_map_count + 32 - 1) / 32 * 32;
			}
			else
			{
				int threadblock_count = (input_feature_map_count + 256 - 1) / 256;
				threadblock_size = (input_feature_map_count + threadblock_count - 1) / threadblock_count;
				threadblock_size = (threadblock_size + 32 - 1) / 32 * 32;
			}

			return threadblock_size;
		}
	}
}
