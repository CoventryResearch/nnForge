#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2016 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "cdf_max_layer_tester_cuda.h"

#include <hip/hip_runtime.h>

#include "util_cuda.h"
#include "neural_network_cuda_exception.h"

#include "../cdf_max_layer.h"
#include <memory>

template<bool IS_MIN>
__global__ void cdf_max_kernel(
	float * __restrict output,
	const float * __restrict input,
	int neuron_count,
	int entry_subsampling_size,
	int output_entry_count)
{
	int neuron_id = blockIdx.x * blockDim.x + threadIdx.x;
	int output_entry_id = blockIdx.y * blockDim.y + threadIdx.y;

	if ((neuron_id < neuron_count) && (output_entry_id < output_entry_count))
	{
		int input_offset = output_entry_id * neuron_count * entry_subsampling_size + neuron_id;
		float product = 1.0F;
		#pragma unroll 4
		for(int i = 0; i < entry_subsampling_size; ++i)
		{
			float val = input[input_offset];
			if (IS_MIN)
				product *= (1.0F - val);
			else
				product *= val;
			input_offset += neuron_count;
		}
		if (IS_MIN)
			product = 1.0F - product;
		output[output_entry_id * neuron_count + neuron_id] = product;
	}
}

namespace nnforge
{
	namespace cuda
	{
		void cdf_max_layer_tester_cuda::enqueue_forward_propagation(
			hipStream_t stream_id,
			cuda_linear_buffer_device::ptr output_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data_custom,
			const std::vector<cuda_linear_buffer_device::const_ptr>& input_buffers,
			const std::vector<cuda_linear_buffer_device::const_ptr>& persistent_working_data,
			cuda_linear_buffer_device::ptr temporary_working_fixed_buffer,
			cuda_linear_buffer_device::ptr temporary_working_per_entry_buffer,
			unsigned int entry_count)
		{
			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				output_elem_count_per_entry,
				entry_count,
				1);

			if (is_min)
				cdf_max_kernel<true><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
					*output_buffer,
					*input_buffers[0],
					output_elem_count_per_entry,
					entry_subsampling_size,
					entry_count);
			else
				cdf_max_kernel<false><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
					*output_buffer,
					*input_buffers[0],
					output_elem_count_per_entry,
					entry_subsampling_size,
					entry_count);
		}

		void cdf_max_layer_tester_cuda::tester_configured()
		{
			std::shared_ptr<const cdf_max_layer> layer_derived = std::dynamic_pointer_cast<const cdf_max_layer>(layer_schema);

			entry_subsampling_size = layer_derived->entry_subsampling_size;
			is_min = layer_derived->is_min;
		}
	}
}
