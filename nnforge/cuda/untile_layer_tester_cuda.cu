#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2015 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "untile_layer_tester_cuda.h"

#include <hip/hip_runtime.h>
#include <boost/format.hpp>

#include "util_cuda.h"
#include "neural_network_cuda_exception.h"

#include "../untile_layer.h"
#include "../nn_types.h"

__global__ void untile_kernel(
	float * __restrict output,
	const float * __restrict input,
	const int * __restrict output_positions,
	const int * __restrict output_offsets,
	int neuron_count_per_input_feature_map,
	int neuron_count_per_output_feature_map,
	int feature_map_count,
	int output_entry_count,
	int_fastdiv local_entry_count)
{
	int input_neuron_output_local_entry_pair_id = blockIdx.x * blockDim.x + threadIdx.x;
	int feature_map_id = blockIdx.y * blockDim.y + threadIdx.y;
	int output_entry_id = blockIdx.z * blockDim.z + threadIdx.z;

	int input_neuron_id = input_neuron_output_local_entry_pair_id / local_entry_count;
	int local_entry_id = input_neuron_output_local_entry_pair_id - input_neuron_id * local_entry_count;

	bool b_valid = (input_neuron_id < neuron_count_per_input_feature_map) && (local_entry_id < local_entry_count) && (feature_map_id < feature_map_count) && (output_entry_id < output_entry_count);
	if (b_valid)
	{
		int input_entry_id = output_entry_id * local_entry_count + local_entry_id;
		int input_offset = (input_entry_id * feature_map_count + feature_map_id) * neuron_count_per_input_feature_map + input_neuron_id;
		int output_neuron_offset = __load_nc(output_positions + input_neuron_id) + __load_nc(output_offsets + local_entry_id);
		float val = __load_nc(input + input_offset);
		int output_offset = (output_entry_id * feature_map_count + feature_map_id) * neuron_count_per_output_feature_map + output_neuron_offset;
		output[output_offset] = val;
	}
}

namespace nnforge
{
	namespace cuda
	{
		untile_layer_tester_cuda::untile_layer_tester_cuda()
		{
		}

		untile_layer_tester_cuda::~untile_layer_tester_cuda()
		{
		}

		void untile_layer_tester_cuda::enqueue_test(
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& data,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& data_custom,
			cuda_linear_buffer_device_smart_ptr input_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			unsigned int entry_count)
		{
			const float * input = *input_buffer;
			float * output = *additional_buffers[0];
			const int * output_positions = *additional_buffers[1];
			const int * output_offsets = *additional_buffers[2];

			if (entry_count % total_tiling_factor != 0)
				throw neural_network_exception((boost::format("untile_layer_tester_cuda: entry_count (%1%) is not evenly divisible by total_tiling_factor (%2%)") % entry_count % (int)total_tiling_factor).str());
			int output_entry_count = entry_count / total_tiling_factor;

			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				input_elem_count_per_feature_map * total_tiling_factor,
				output_configuration_specific.feature_map_count,
				output_entry_count);

			untile_kernel<<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
				output,
				input,
				output_positions,
				output_offsets,
				input_elem_count_per_feature_map,
				output_elem_count_per_feature_map,
				output_configuration_specific.feature_map_count,
				output_entry_count,
				total_tiling_factor);
		}

		std::vector<size_t> untile_layer_tester_cuda::get_sizes_of_additional_buffers_per_entry() const
		{
			std::vector<size_t> res;

			res.push_back((output_elem_count_per_entry * sizeof(float) + total_tiling_factor - 1) / (int)total_tiling_factor);

			return res;
		}

		std::vector<size_t> untile_layer_tester_cuda::get_sizes_of_additional_buffers_fixed() const
		{
			std::vector<size_t> res;

			res.push_back(input_elem_count_per_feature_map * sizeof(int));
			res.push_back(total_tiling_factor * sizeof(int));

			return res;
		}

		cuda_linear_buffer_device_smart_ptr untile_layer_tester_cuda::get_output_buffer(
			cuda_linear_buffer_device_smart_ptr input_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers)
		{
			return additional_buffers[0];
		}

		void untile_layer_tester_cuda::tester_configured()
		{
			nnforge_shared_ptr<const untile_layer> layer_derived = nnforge_dynamic_pointer_cast<const untile_layer>(layer_schema);

			upsampling_sizes_list = layer_derived->upsampling_sizes_list;
			total_tiling_factor = layer_derived->get_tiling_factor().get_inverse();
		}

		void untile_layer_tester_cuda::fill_additional_buffers(const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers) const
		{
			{
				std::vector<int> position_list(input_elem_count_per_feature_map);
				{
					std::vector<unsigned int> tiling_sizes(input_configuration_specific.dimension_sizes.size(), 1);
					for(int i = 0; i < upsampling_sizes_list.size(); ++i)
					{
						const std::vector<unsigned int>& upsampling_sizes = upsampling_sizes_list[i];
						for(int j = 0; j < upsampling_sizes.size(); ++j)
							tiling_sizes[j] *= upsampling_sizes[j];
					}

					std::vector<unsigned int> spatial_pos(input_configuration_specific.dimension_sizes.size(), 0);
					for(unsigned int i = 0; i < input_elem_count_per_feature_map; ++i)
					{
						int pos = spatial_pos.back() * tiling_sizes.back();
						for(int j = static_cast<int>(spatial_pos.size() - 2); j >= 0; --j)
							pos = pos * output_configuration_specific.dimension_sizes[j] + spatial_pos[j] * tiling_sizes[j];
						position_list[i] = pos;

						for(int j = 0; j < spatial_pos.size(); ++j)
						{
							if ((++spatial_pos[j]) < input_configuration_specific.dimension_sizes[j])
								break;
							spatial_pos[j] = 0;
						}
					}
				}
				cuda_safe_call(hipMemcpy(*additional_buffers[1], &(*position_list.begin()), sizeof(int) * position_list.size(), hipMemcpyHostToDevice));
			}

			{
				std::vector<int> offset_list(total_tiling_factor);
				{
					std::vector<std::vector<unsigned int> > positions_list;
					positions_list.push_back(std::vector<unsigned int>(output_configuration_specific.dimension_sizes.size(), 0));

					std::vector<unsigned int> total_upsampling_sizes(upsampling_sizes_list.front().size(), 1);

					for(int level = static_cast<unsigned int>(upsampling_sizes_list.size()) - 1; level >= 0; --level)
					{
						std::vector<std::vector<unsigned int> > new_positions_list;
						const std::vector<unsigned int>& upsampling_sizes = upsampling_sizes_list[level];

						unsigned int local_tiling_count = 1;
						for(std::vector<unsigned int>::const_iterator it = upsampling_sizes.begin(); it != upsampling_sizes.end(); ++it)
							local_tiling_count *= *it;

						for(std::vector<std::vector<unsigned int> >::const_iterator it = positions_list.begin(); it != positions_list.end(); ++it)
						{
							const std::vector<unsigned int>& current_positions = *it;

							std::vector<unsigned int> local_pos(upsampling_sizes.size(), 0);
							for(unsigned int i = 0; i < local_tiling_count; ++i)
							{
								std::vector<unsigned int> new_untiled_positions(current_positions);
								for(unsigned int j = 0; j < static_cast<unsigned int>(upsampling_sizes.size()); ++j)
									new_untiled_positions[j] += local_pos[j] * total_upsampling_sizes[j];

								new_positions_list.push_back(new_untiled_positions);

								for(int j = 0; j < local_pos.size(); ++j)
								{
									if ((++local_pos[j]) < upsampling_sizes[j])
										break;
									local_pos[j] = 0;
								}
							}
						}

						for(unsigned int i = 0; i < static_cast<unsigned int>(total_upsampling_sizes.size()); ++i)
							total_upsampling_sizes[i] *= upsampling_sizes[i];

						positions_list = new_positions_list;
					}

					for(unsigned int i = 0; i < total_tiling_factor; ++i)
					{
						const std::vector<unsigned int>& positions = positions_list[i];
						int pos = positions.back();
						for(int j = static_cast<int>(positions.size() - 2); j >= 0; --j)
							pos = pos * output_configuration_specific.dimension_sizes[j] + positions[j];
						offset_list[i] = pos;
					}
				}
				cuda_safe_call(hipMemcpy(*additional_buffers[2], &(*offset_list.begin()), sizeof(int) * offset_list.size(), hipMemcpyHostToDevice));
			}
		}
	}
}
