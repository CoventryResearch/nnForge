#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2016 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "local_contrast_subtractive_2d_layer_updater_cuda.h"

#include "../local_contrast_subtractive_layer.h"
#include "../neural_network_exception.h"

#include "util_cuda.h"

#include <memory>

namespace nnforge
{
	namespace cuda
	{
		__global__ void local_contrast_subtractive_2d_blur_horizontal_upd_kernel(
			const float * __restrict input,
			float * __restrict output,
			const unsigned int * __restrict affected_feature_map_list,
			const float * __restrict weights,
			int input_feature_map_count,
			int affected_feature_map_count,
			int window_width,
			int width,
			int height,
			int entry_count)
		{
			int x = blockIdx.x * blockDim.x + threadIdx.x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;
			int z = blockIdx.z * blockDim.z + threadIdx.z;
			int entry_id = z / affected_feature_map_count;

			bool in_bounds = (x < width) && (y < height) && (entry_id < entry_count);
			if (in_bounds)
			{
				int affected_feature_map_index = z - (entry_id * affected_feature_map_count);
				int affected_feature_map_id = affected_feature_map_list[affected_feature_map_index];
				const float * current_input = input + (int)(((entry_id * input_feature_map_count + affected_feature_map_id) * height + y) * width + x);
				const float * current_input_low = current_input;
				const float * current_input_high = current_input;
				const float * current_weights = weights;
				float res = *current_input * *current_weights;
				#pragma unroll 4
				for(int i = 1; i < window_width; ++i)
				{
					current_weights++;
					if (i < x + 1)
						current_input_low--;
					if (i > x + 1)
						current_input_low++;
					if (i < width - x)
						current_input_high++;
					if (i > width - x)
						current_input_high--;
					res += (*current_input_low + *current_input_high) * *current_weights;
				}

				output[(z * height + y) * width + x] = res;
			}
		}

		template<int WINDOW_WIDTH>
		__global__ void local_contrast_subtractive_2d_blur_horizontal_exact_upd_kernel(
			const float * __restrict input,
			float * __restrict output,
			const unsigned int * __restrict affected_feature_map_list,
			const float * __restrict weights,
			int input_feature_map_count,
			int affected_feature_map_count,
			int width,
			int height,
			int entry_count)
		{
			int x = blockIdx.x * blockDim.x + threadIdx.x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;
			int z = blockIdx.z * blockDim.z + threadIdx.z;
			int entry_id = z / affected_feature_map_count;

			bool in_bounds = (x < width) && (y < height) && (entry_id < entry_count);
			if (in_bounds)
			{
				int affected_feature_map_index = z - (entry_id * affected_feature_map_count);
				int affected_feature_map_id = affected_feature_map_list[affected_feature_map_index];
				const float * current_input = input + (int)(((entry_id * input_feature_map_count + affected_feature_map_id) * height + y) * width + x);
				const float * current_input_low = current_input;
				const float * current_input_high = current_input;
				const float * current_weights = weights;
				float res = *current_input * *current_weights;
				#pragma unroll
				for(int i = 1; i < WINDOW_WIDTH; ++i)
				{
					current_weights++;
					if (i < x + 1)
						current_input_low--;
					if (i > x + 1)
						current_input_low++;
					if (i < width - x)
						current_input_high++;
					if (i > width - x)
						current_input_high--;
					res += (*current_input_low + *current_input_high) * *current_weights;
				}

				output[(z * height + y) * width + x] = res;
			}
		}

		__global__ void local_contrast_subtractive_2d_blur_vertical_and_subtract_upd_kernel(
			const float * __restrict input,
			const float * __restrict original_input,
			float * __restrict output,
			const unsigned int * __restrict affected_feature_map_list,
			const float * __restrict weights,
			int input_feature_map_count,
			int affected_feature_map_count,
			int window_height,
			int width,
			int height,
			int entry_count)
		{
			int x = blockIdx.x * blockDim.x + threadIdx.x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;
			int z = blockIdx.z * blockDim.z + threadIdx.z;
			int entry_id = z / affected_feature_map_count;

			bool in_bounds = (x < width) && (y < height) && (entry_id < entry_count);
			if (in_bounds)
			{
				int affected_feature_map_index = z - (entry_id * affected_feature_map_count);
				int affected_feature_map_id = affected_feature_map_list[affected_feature_map_index];
				const float * current_input = input + (int)((z * height + y) * width + x);
				const float * current_input_low = current_input;
				const float * current_input_high = current_input;
				const float * current_weights = weights;
				float res = *current_input * *current_weights;
				#pragma unroll 4
				for(int i = 1; i < window_height; ++i)
				{
					current_weights++;
					if (i < y + 1)
						current_input_low -= width;
					if (i > y + 1)
						current_input_low += width;
					if (i < height - y)
						current_input_high += width;
					if (i > height - y)
						current_input_high -= width;
					res += (*current_input_low + *current_input_high) * *current_weights;
				}

				int offset = ((entry_id * input_feature_map_count + affected_feature_map_id) * height + y) * width + x;
				output[offset] = original_input[offset] - res;
			}
		}

		template<int WINDOW_HEIGHT>
		__global__ void local_contrast_subtractive_2d_blur_vertical_and_subtract_exact_upd_kernel(
			const float * __restrict input,
			const float * __restrict original_input,
			float * __restrict output,
			const unsigned int * __restrict affected_feature_map_list,
			const float * __restrict weights,
			int input_feature_map_count,
			int affected_feature_map_count,
			int width,
			int height,
			int entry_count)
		{
			int x = blockIdx.x * blockDim.x + threadIdx.x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;
			int z = blockIdx.z * blockDim.z + threadIdx.z;
			int entry_id = z / affected_feature_map_count;

			bool in_bounds = (x < width) && (y < height) && (entry_id < entry_count);
			if (in_bounds)
			{
				int affected_feature_map_index = z - (entry_id * affected_feature_map_count);
				int affected_feature_map_id = affected_feature_map_list[affected_feature_map_index];
				const float * current_input = input + (int)((z * height + y) * width + x);
				const float * current_input_low = current_input;
				const float * current_input_high = current_input;
				const float * current_weights = weights;
				float res = *current_input * *current_weights;
				#pragma unroll
				for(int i = 1; i < WINDOW_HEIGHT; ++i)
				{
					current_weights++;
					if (i < y + 1)
						current_input_low -= width;
					if (i > y + 1)
						current_input_low += width;
					if (i < height - y)
						current_input_high += width;
					if (i > height - y)
						current_input_high -= width;
					res += (*current_input_low + *current_input_high) * *current_weights;
				}

				int offset = ((entry_id * input_feature_map_count + affected_feature_map_id) * height + y) * width + x;
				output[offset] = original_input[offset] - res;
			}
		}

		__global__ void local_contrast_subtractive_2d_copy_unaffected_upd_kernel(
			const float * __restrict original_input,
			float * __restrict output,
			const unsigned int * __restrict unaffected_feature_map_list,
			int input_feature_map_count,
			int unaffected_feature_map_count,
			int elem_count_per_fature_map,
			int entry_count)
		{
			int elem_id = blockIdx.x * blockDim.x + threadIdx.x;
			int unaffected_feature_map_index = blockIdx.y * blockDim.y + threadIdx.y;
			int entry_id = blockIdx.z * blockDim.z + threadIdx.z;
			bool in_bounds = (elem_id < elem_count_per_fature_map) && (unaffected_feature_map_index < unaffected_feature_map_count) && (entry_id < entry_count);
			if (in_bounds)
			{
				int unaffected_feature_map_id = unaffected_feature_map_list[unaffected_feature_map_index];
				int offset = (entry_id * input_feature_map_count + unaffected_feature_map_id) * elem_count_per_fature_map + elem_id;
				output[offset] = original_input[offset];
			}
		}

		void local_contrast_subtractive_2d_layer_updater_cuda::enqueue_forward_propagation(
			hipStream_t stream_id,
			cuda_linear_buffer_device::ptr output_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data_custom,
			const std::vector<cuda_linear_buffer_device::const_ptr>& input_buffers,
			const std::vector<cuda_linear_buffer_device::const_ptr>& persistent_working_data,
			cuda_linear_buffer_device::ptr temporary_working_fixed_buffer,
			cuda_linear_buffer_device::ptr temporary_working_per_entry_buffer,
			cuda_linear_buffer_device::ptr temporary_fixed_buffer,
			cuda_linear_buffer_device::ptr temporary_per_entry_buffer,
			unsigned int entry_count)
		{
			std::pair<dim3, dim3> kernel_1st_dims = cuda_util::get_grid_and_threadblock_sizes_2d_access(
				*cuda_config,
				output_configuration_specific.dimension_sizes[0],
				output_configuration_specific.dimension_sizes[1],
				affected_feature_map_count * entry_count);
			switch(half_window_sizes[0])
			{
			case 1:
				local_contrast_subtractive_2d_blur_horizontal_exact_upd_kernel<1><<<kernel_1st_dims.first, kernel_1st_dims.second, 0, stream_id>>>(*input_buffers[0], *temporary_working_per_entry_buffer, *schema_data[0], *schema_data[1], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 2:
				local_contrast_subtractive_2d_blur_horizontal_exact_upd_kernel<2><<<kernel_1st_dims.first, kernel_1st_dims.second, 0, stream_id>>>(*input_buffers[0], *temporary_working_per_entry_buffer, *schema_data[0], *schema_data[1], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 3:
				local_contrast_subtractive_2d_blur_horizontal_exact_upd_kernel<3><<<kernel_1st_dims.first, kernel_1st_dims.second, 0, stream_id>>>(*input_buffers[0], *temporary_working_per_entry_buffer, *schema_data[0], *schema_data[1], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 4:
				local_contrast_subtractive_2d_blur_horizontal_exact_upd_kernel<4><<<kernel_1st_dims.first, kernel_1st_dims.second, 0, stream_id>>>(*input_buffers[0], *temporary_working_per_entry_buffer, *schema_data[0], *schema_data[1], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 5:
				local_contrast_subtractive_2d_blur_horizontal_exact_upd_kernel<5><<<kernel_1st_dims.first, kernel_1st_dims.second, 0, stream_id>>>(*input_buffers[0], *temporary_working_per_entry_buffer, *schema_data[0], *schema_data[1], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 6:
				local_contrast_subtractive_2d_blur_horizontal_exact_upd_kernel<6><<<kernel_1st_dims.first, kernel_1st_dims.second, 0, stream_id>>>(*input_buffers[0], *temporary_working_per_entry_buffer, *schema_data[0], *schema_data[1], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 7:
				local_contrast_subtractive_2d_blur_horizontal_exact_upd_kernel<7><<<kernel_1st_dims.first, kernel_1st_dims.second, 0, stream_id>>>(*input_buffers[0], *temporary_working_per_entry_buffer, *schema_data[0], *schema_data[1], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 8:
				local_contrast_subtractive_2d_blur_horizontal_exact_upd_kernel<8><<<kernel_1st_dims.first, kernel_1st_dims.second, 0, stream_id>>>(*input_buffers[0], *temporary_working_per_entry_buffer, *schema_data[0], *schema_data[1], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 9:
				local_contrast_subtractive_2d_blur_horizontal_exact_upd_kernel<9><<<kernel_1st_dims.first, kernel_1st_dims.second, 0, stream_id>>>(*input_buffers[0], *temporary_working_per_entry_buffer, *schema_data[0], *schema_data[1], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 10:
				local_contrast_subtractive_2d_blur_horizontal_exact_upd_kernel<10><<<kernel_1st_dims.first, kernel_1st_dims.second, 0, stream_id>>>(*input_buffers[0], *temporary_working_per_entry_buffer, *schema_data[0], *schema_data[1], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			default:
				local_contrast_subtractive_2d_blur_horizontal_upd_kernel<<<kernel_1st_dims.first, kernel_1st_dims.second, 0, stream_id>>>(
					*input_buffers[0],
					*temporary_working_per_entry_buffer,
					*schema_data[0],
					*schema_data[1],
					output_configuration_specific.feature_map_count,
					affected_feature_map_count,
					half_window_sizes[0],
					output_configuration_specific.dimension_sizes[0],
					output_configuration_specific.dimension_sizes[1],
					entry_count);
				break;
			}

			std::pair<dim3, dim3> kernel_2nd_dims = cuda_util::get_grid_and_threadblock_sizes_2d_access(
				*cuda_config,
				output_configuration_specific.dimension_sizes[0],
				output_configuration_specific.dimension_sizes[1],
				affected_feature_map_count * entry_count);
			switch(half_window_sizes[1])
			{
			case 1:
				local_contrast_subtractive_2d_blur_vertical_and_subtract_exact_upd_kernel<1><<<kernel_2nd_dims.first, kernel_2nd_dims.second, 0, stream_id>>>(*temporary_working_per_entry_buffer, *input_buffers[0], *output_buffer, *schema_data[0], *schema_data[2], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 2:
				local_contrast_subtractive_2d_blur_vertical_and_subtract_exact_upd_kernel<2><<<kernel_2nd_dims.first, kernel_2nd_dims.second, 0, stream_id>>>(*temporary_working_per_entry_buffer, *input_buffers[0], *output_buffer, *schema_data[0], *schema_data[2], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 3:
				local_contrast_subtractive_2d_blur_vertical_and_subtract_exact_upd_kernel<3><<<kernel_2nd_dims.first, kernel_2nd_dims.second, 0, stream_id>>>(*temporary_working_per_entry_buffer, *input_buffers[0], *output_buffer, *schema_data[0], *schema_data[2], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 4:
				local_contrast_subtractive_2d_blur_vertical_and_subtract_exact_upd_kernel<4><<<kernel_2nd_dims.first, kernel_2nd_dims.second, 0, stream_id>>>(*temporary_working_per_entry_buffer, *input_buffers[0], *output_buffer, *schema_data[0], *schema_data[2], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 5:
				local_contrast_subtractive_2d_blur_vertical_and_subtract_exact_upd_kernel<5><<<kernel_2nd_dims.first, kernel_2nd_dims.second, 0, stream_id>>>(*temporary_working_per_entry_buffer, *input_buffers[0], *output_buffer, *schema_data[0], *schema_data[2], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 6:
				local_contrast_subtractive_2d_blur_vertical_and_subtract_exact_upd_kernel<6><<<kernel_2nd_dims.first, kernel_2nd_dims.second, 0, stream_id>>>(*temporary_working_per_entry_buffer, *input_buffers[0], *output_buffer, *schema_data[0], *schema_data[2], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 7:
				local_contrast_subtractive_2d_blur_vertical_and_subtract_exact_upd_kernel<7><<<kernel_2nd_dims.first, kernel_2nd_dims.second, 0, stream_id>>>(*temporary_working_per_entry_buffer, *input_buffers[0], *output_buffer, *schema_data[0], *schema_data[2], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 8:
				local_contrast_subtractive_2d_blur_vertical_and_subtract_exact_upd_kernel<8><<<kernel_2nd_dims.first, kernel_2nd_dims.second, 0, stream_id>>>(*temporary_working_per_entry_buffer, *input_buffers[0], *output_buffer, *schema_data[0], *schema_data[2], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 9:
				local_contrast_subtractive_2d_blur_vertical_and_subtract_exact_upd_kernel<9><<<kernel_2nd_dims.first, kernel_2nd_dims.second, 0, stream_id>>>(*temporary_working_per_entry_buffer, *input_buffers[0], *output_buffer, *schema_data[0], *schema_data[2], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			case 10:
				local_contrast_subtractive_2d_blur_vertical_and_subtract_exact_upd_kernel<10><<<kernel_2nd_dims.first, kernel_2nd_dims.second, 0, stream_id>>>(*temporary_working_per_entry_buffer, *input_buffers[0], *output_buffer, *schema_data[0], *schema_data[2], output_configuration_specific.feature_map_count, affected_feature_map_count, output_configuration_specific.dimension_sizes[0], output_configuration_specific.dimension_sizes[1], entry_count);
				break;
			default:
				local_contrast_subtractive_2d_blur_vertical_and_subtract_upd_kernel<<<kernel_2nd_dims.first, kernel_2nd_dims.second, 0, stream_id>>>(
					*temporary_working_per_entry_buffer,
					*input_buffers[0],
					*output_buffer,
					*schema_data[0],
					*schema_data[2],
					output_configuration_specific.feature_map_count,
					affected_feature_map_count,
					half_window_sizes[1],
					output_configuration_specific.dimension_sizes[0],
					output_configuration_specific.dimension_sizes[1],
					entry_count);
				break;
			}

			if ((unaffected_feature_map_count > 0) && ((const float *)*input_buffers[0] != (const float *)*output_buffer))
			{
				std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_2d_access(
					*cuda_config,
					output_elem_count_per_feature_map,
					unaffected_feature_map_count,
					entry_count);
				local_contrast_subtractive_2d_copy_unaffected_upd_kernel<<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
					*input_buffers[0],
					*output_buffer,
					*schema_data[3],
					output_configuration_specific.feature_map_count,
					unaffected_feature_map_count,
					output_elem_count_per_feature_map,
					entry_count);
			}
		}

		void local_contrast_subtractive_2d_layer_updater_cuda::updater_configured()
		{
			std::shared_ptr<const local_contrast_subtractive_layer> layer_derived = std::dynamic_pointer_cast<const local_contrast_subtractive_layer>(layer_schema);

			affected_feature_map_count = static_cast<int>(layer_derived->feature_maps_affected.size());
			unaffected_feature_map_count = static_cast<int>(layer_derived->feature_maps_unaffected.size());

			for(std::vector<unsigned int>::const_iterator it = layer_derived->window_sizes.begin(); it != layer_derived->window_sizes.end(); ++it)
				half_window_sizes.push_back(static_cast<int>((*it + 1) >> 1));

			central_mult = 1.0F - (2.0F * layer_derived->window_weights_list[0][0] * layer_derived->window_weights_list[1][0]);
		}

		size_t local_contrast_subtractive_2d_layer_updater_cuda::get_temporary_working_per_entry_buffer_size(const layer_action& action) const
		{
			return output_elem_count_per_feature_map * affected_feature_map_count * sizeof(float);
		}

		int local_contrast_subtractive_2d_layer_updater_cuda::get_input_index_layer_can_write(const layer_action& action) const
		{
			return 0;
		}
	}
}
