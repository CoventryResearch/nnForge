#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2015 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "concat_layer_updater_cuda.h"

#include "util_cuda.h"
#include "neural_network_cuda_exception.h"

namespace nnforge
{
	namespace cuda
	{
		__global__ void concat_backprop_upd_kernel(
			float * __restrict input_errors,
			const float * __restrict output_errors,
			int elem_count) 
		{
			int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
			if (elem_id < elem_count)
			{
				input_errors[elem_id] += output_errors[elem_id];
			}
		}

		concat_layer_updater_cuda::concat_layer_updater_cuda()
		{
		}

		concat_layer_updater_cuda::~concat_layer_updater_cuda()
		{
		}

		void concat_layer_updater_cuda::enqueue_forward_propagation(
			hipStream_t stream_id,
			cuda_linear_buffer_device::ptr output_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device::ptr>& data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data_custom,
			const std::vector<cuda_linear_buffer_device::const_ptr>& input_buffers,
			const std::vector<cuda_linear_buffer_device::const_ptr>& persistent_working_data,
			cuda_linear_buffer_device::ptr temporary_working_fixed_buffer,
			cuda_linear_buffer_device::ptr temporary_working_per_entry_buffer,
			cuda_linear_buffer_device::ptr temporary_per_entry_buffer,
			unsigned int entry_count)
		{
			unsigned int offset = 0;
			for(unsigned int i = 0; i < static_cast<unsigned int>(input_configuration_specific_list.size()); ++i)
			{
				unsigned int elem_count = input_elem_count_per_entry_list[i] * entry_count;

				if ((offset & 3) == 0)
					cuda_util::copy_buffer(
						*cuda_config,
						*input_buffers[i],
						(float *)(*output_buffer) + offset,
						elem_count,
						stream_id);
				else
					cuda_safe_call(hipMemcpyAsync(
						(float *)(*output_buffer) + offset,
						*input_buffers[i],
						elem_count * sizeof(float),
						hipMemcpyDeviceToDevice,
						stream_id));

				offset += elem_count;
			}
		}

		void concat_layer_updater_cuda::enqueue_backward_data_propagation(
			hipStream_t stream_id,
			unsigned int input_index,
			cuda_linear_buffer_device::ptr input_errors_buffer,
			cuda_linear_buffer_device::const_ptr output_errors_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device::ptr>& data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data_custom,
			const std::vector<cuda_linear_buffer_device::const_ptr>& input_neurons_buffers,
			cuda_linear_buffer_device::const_ptr output_neurons_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& persistent_working_data,
			cuda_linear_buffer_device::ptr temporary_working_fixed_buffer,
			cuda_linear_buffer_device::ptr temporary_working_per_entry_buffer,
			cuda_linear_buffer_device::const_ptr temporary_per_entry_buffer,
			bool add_update_to_destination,
			unsigned int entry_count)
		{
			unsigned int offset = 0;
			for(unsigned int i = 0; i < input_index; ++i)
			{
				unsigned int elem_count = input_elem_count_per_entry_list[i] * entry_count;
				offset += elem_count;
			}
			unsigned int elem_count = input_elem_count_per_entry_list[input_index] * entry_count;

			if (add_update_to_destination)
			{
				std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
					*cuda_config,
					elem_count);
				concat_backprop_upd_kernel<<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
					*input_errors_buffer,
					(const float *)(*output_errors_buffer) + offset,
					elem_count);
			}
			else
			{
				if ((offset & 3) == 0)
					cuda_util::copy_buffer(
						*cuda_config,
						(const float *)(*output_errors_buffer) + offset,
						*input_errors_buffer,
						elem_count,
						stream_id);
				else
					cuda_safe_call(hipMemcpyAsync(
						*input_errors_buffer,
						(const float *)(*output_errors_buffer) + offset,
						elem_count * sizeof(float),
						hipMemcpyDeviceToDevice,
						stream_id));
			}
		}

		bool concat_layer_updater_cuda::is_backward_data_dependent_on_input_buffer(unsigned int action_input_index, unsigned int data_input_index) const
		{
			return false;
		}

		bool concat_layer_updater_cuda::is_backward_data_dependent_on_output_buffer(unsigned int action_input_index) const
		{
			return false;
		}
	}
}
