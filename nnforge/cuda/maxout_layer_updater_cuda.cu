#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2015 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "maxout_layer_updater_cuda.h"

#include <hip/hip_runtime.h>

#include "util_cuda.h"
#include "neural_network_cuda_exception.h"

#include "../maxout_layer.h"
#include "../nn_types.h"

__global__ void maxout_upd_kernel(
	float * __restrict output,
	int * __restrict max_feature_map_positions,
	const float * __restrict input,
	int neuron_count_per_feature_map,
	int input_feature_map_count,
	int output_feature_map_count,
	int feature_map_subsampling_size,
	int entry_count)
{
	int neuron_id = blockIdx.x * blockDim.x + threadIdx.x;
	int output_feature_map_id = blockIdx.y * blockDim.y + threadIdx.y;
	int entry_id = blockIdx.z * blockDim.z + threadIdx.z;

	if ((neuron_id < neuron_count_per_feature_map) && (output_feature_map_id < output_feature_map_count) && (entry_id < entry_count))
	{
		int input_offset = (entry_id * input_feature_map_count + output_feature_map_id) * neuron_count_per_feature_map + neuron_id;
		float max_val = input[input_offset];
		int max_pos = 0;
		for(int i = 1; i < feature_map_subsampling_size; ++i)
		{
			input_offset += output_feature_map_count * neuron_count_per_feature_map;
			float new_val = input[input_offset];
			if (new_val > max_val)
			{
				max_val = new_val;
				max_pos = i;
			}
		}
		int output_offset = (entry_id * output_feature_map_count + output_feature_map_id) * neuron_count_per_feature_map + neuron_id;
		output[output_offset] = max_val;
		max_feature_map_positions[output_offset] = max_pos;
	}
}

template<bool add_update_to_destination>
__global__ void maxout_deriviative_upd_kernel(
	float * __restrict input_errors,
	const int * __restrict max_feature_map_positions,
	const float * __restrict output_errors,
	int neuron_count_per_feature_map,
	int input_feature_map_count,
	int output_feature_map_count,
	int feature_map_subsampling_size,
	int entry_count)
{
	int neuron_id = blockIdx.x * blockDim.x + threadIdx.x;
	int output_feature_map_id = blockIdx.y * blockDim.y + threadIdx.y;
	int entry_id = blockIdx.z * blockDim.z + threadIdx.z;

	if ((neuron_id < neuron_count_per_feature_map) && (output_feature_map_id < output_feature_map_count) && (entry_id < entry_count))
	{
		int output_offset = (entry_id * output_feature_map_count + output_feature_map_id) * neuron_count_per_feature_map + neuron_id;
		int max_feature_map = max_feature_map_positions[output_offset];
		float output_error = output_errors[output_offset];

		int input_offset = (entry_id * input_feature_map_count + output_feature_map_id) * neuron_count_per_feature_map + neuron_id;
		for(int i = 0; i < feature_map_subsampling_size; ++i)
		{
			if (add_update_to_destination)
				input_errors[input_offset] += ((i == max_feature_map) ? output_error : 0.0F);
			else
				input_errors[input_offset] = ((i == max_feature_map) ? output_error : 0.0F);
			input_offset += output_feature_map_count * neuron_count_per_feature_map;
		}
	}
}

namespace nnforge
{
	namespace cuda
	{
		maxout_layer_updater_cuda::maxout_layer_updater_cuda()
		{
		}

		maxout_layer_updater_cuda::~maxout_layer_updater_cuda()
		{
		}

		void maxout_layer_updater_cuda::enqueue_forward_propagation(
			hipStream_t stream_id,
			cuda_linear_buffer_device::ptr output_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device::ptr>& data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data_custom,
			const std::vector<cuda_linear_buffer_device::const_ptr>& input_buffers,
			const std::vector<cuda_linear_buffer_device::const_ptr>& persistent_working_data,
			cuda_linear_buffer_device::ptr temporary_working_fixed_buffer,
			cuda_linear_buffer_device::ptr temporary_working_per_entry_buffer,
			cuda_linear_buffer_device::ptr temporary_per_entry_buffer,
			unsigned int entry_count)
		{
			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				output_elem_count_per_feature_map,
				output_configuration_specific.feature_map_count,
				entry_count);

			maxout_upd_kernel<<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
				*output_buffer,
				*temporary_per_entry_buffer,
				*input_buffers[0],
				output_elem_count_per_feature_map,
				input_configuration_specific_list[0].feature_map_count,
				output_configuration_specific.feature_map_count,
				feature_map_subsampling_size,
				entry_count);
		}

		void maxout_layer_updater_cuda::enqueue_backward_data_propagation(
			hipStream_t stream_id,
			unsigned int input_index,
			cuda_linear_buffer_device::ptr input_errors_buffer,
			cuda_linear_buffer_device::const_ptr output_errors_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device::ptr>& data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data_custom,
			const std::vector<cuda_linear_buffer_device::const_ptr>& input_neurons_buffers,
			cuda_linear_buffer_device::const_ptr output_neurons_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& persistent_working_data,
			cuda_linear_buffer_device::ptr temporary_working_fixed_buffer,
			cuda_linear_buffer_device::ptr temporary_working_per_entry_buffer,
			cuda_linear_buffer_device::const_ptr temporary_per_entry_buffer,
			bool add_update_to_destination,
			unsigned int entry_count)
		{
			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				output_elem_count_per_feature_map,
				output_configuration_specific.feature_map_count,
				entry_count);

			if (add_update_to_destination)
				maxout_deriviative_upd_kernel<true><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
					*input_errors_buffer,
					*temporary_per_entry_buffer,
					*output_errors_buffer,
					output_elem_count_per_feature_map,
					input_configuration_specific_list[0].feature_map_count,
					output_configuration_specific.feature_map_count,
					feature_map_subsampling_size,
					entry_count);
			else
				maxout_deriviative_upd_kernel<false><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
					*input_errors_buffer,
					*temporary_per_entry_buffer,
					*output_errors_buffer,
					output_elem_count_per_feature_map,
					input_configuration_specific_list[0].feature_map_count,
					output_configuration_specific.feature_map_count,
					feature_map_subsampling_size,
					entry_count);
		}

		bool maxout_layer_updater_cuda::is_backward_data_dependent_on_input_buffer(unsigned int action_input_index, unsigned int data_input_index) const
		{
			return false;
		}

		bool maxout_layer_updater_cuda::is_backward_data_dependent_on_output_buffer(unsigned int action_input_index) const
		{
			return false;
		}

		bool maxout_layer_updater_cuda::is_backward_data_dependent_on_temporary_per_entry_buffer(unsigned int action_input_index) const
		{
			return true;
		}

		void maxout_layer_updater_cuda::updater_configured()
		{
			nnforge_shared_ptr<const maxout_layer> layer_derived = nnforge_dynamic_pointer_cast<const maxout_layer>(layer_schema);

			feature_map_subsampling_size = layer_derived->feature_map_subsampling_size;
		}

		size_t maxout_layer_updater_cuda::get_temporary_per_entry_buffer_size() const
		{
			return output_elem_count_per_entry * sizeof(int);
		}
	}
}
