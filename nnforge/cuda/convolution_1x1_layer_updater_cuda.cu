/*
 *  Copyright 2011-2014 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "convolution_1x1_layer_updater_cuda.h"

#include "util_cuda.h"
#include "neural_network_cublas_exception.h"
#include "neural_network_cuda_exception.h"
#include "neural_network_cudnn_exception.h"

#include "../convolution_layer.h"

namespace nnforge
{
	namespace cuda
	{
		convolution_1x1_layer_updater_cuda::convolution_1x1_layer_updater_cuda()
			: output_data_desc(0)
			, bias_desc(0)
		{
			cudnn_safe_call(hipdnnCreateTensorDescriptor(&output_data_desc));
			cudnn_safe_call(hipdnnCreateTensorDescriptor(&bias_desc));
		}

		convolution_1x1_layer_updater_cuda::~convolution_1x1_layer_updater_cuda()
		{
			hipdnnDestroyTensorDescriptor(output_data_desc);
			hipdnnDestroyTensorDescriptor(bias_desc);
		}

		void convolution_1x1_layer_updater_cuda::enqueue_test(
			unsigned int offset_input_entry_id,
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data_custom,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			cuda_linear_buffer_device_smart_ptr output_neurons_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			std::vector<cuda_memobject_smart_ptr>& dynamic_memobjects,
			unsigned int entry_count,
			bool force_deterministic)
		{
			{
				cuda_util::transpose(
					*cuda_config,
					(const float *)(*input_neurons_buffer) + input_elem_count_per_entry * offset_input_entry_id,
					*additional_buffers[0],
					input_elem_count_per_feature_map,
					input_configuration_specific.feature_map_count,
					entry_count,
					stream_id);

				cublas_safe_call(hipblasSetStream(cuda_config->get_cublas_handle(), stream_id));
				float alpha = 1.0F;
				float beta = 0.0F;
				cublas_safe_call(hipblasSgemm(
					cuda_config->get_cublas_handle(),
					HIPBLAS_OP_T,
					HIPBLAS_OP_N,
					output_configuration_specific.feature_map_count,
					entry_count * input_elem_count_per_feature_map,
					input_configuration_specific.feature_map_count,
					&alpha,
					*data[0],
					input_configuration_specific.feature_map_count,
					*additional_buffers[0],
					input_configuration_specific.feature_map_count,
					&beta,
					*additional_buffers[1],
					output_configuration_specific.feature_map_count));

				cuda_util::transpose(
					*cuda_config,
					*additional_buffers[1],
					*output_neurons_buffer,
					output_configuration_specific.feature_map_count,
					output_elem_count_per_feature_map,
					entry_count,
					stream_id);
			}

			// Add bias
			{
				cudnn_safe_call(hipdnnSetStream(cuda_config->get_cudnn_handle(), stream_id));
				cudnn_safe_call(hipdnnSetTensor4dDescriptor(
					output_data_desc,
					HIPDNN_TENSOR_NCHW,
					HIPDNN_DATA_FLOAT,
					entry_count,
					output_configuration_specific.feature_map_count,
					1,
					output_elem_count_per_feature_map));

				float alpha = 1.0F;
				float beta = 1.0F;
				cudnn_safe_call(hipdnnAddTensor(
					cuda_config->get_cudnn_handle(),
					CUDNN_ADD_SAME_C,
					&alpha,
					bias_desc,
					*data[1],
					&beta,
					output_data_desc,
					*output_neurons_buffer));
			}
		}

		void convolution_1x1_layer_updater_cuda::enqueue_backprop(
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data_custom,
			const_cuda_linear_buffer_device_smart_ptr output_neurons_buffer,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			cuda_linear_buffer_device_smart_ptr output_errors_buffer,
			cuda_linear_buffer_device_smart_ptr input_errors_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			std::vector<cuda_memobject_smart_ptr>& dynamic_memobjects,
			unsigned int entry_count,
			bool force_deterministic)
		{
			cublas_safe_call(hipblasSetStream(cuda_config->get_cublas_handle(), stream_id));
			float alpha = 1.0F;
			float beta = 0.0F;
			cublas_safe_call(hipblasSgemm(
				cuda_config->get_cublas_handle(),
				HIPBLAS_OP_N,
				HIPBLAS_OP_N,
				input_configuration_specific.feature_map_count,
				entry_count * input_elem_count_per_feature_map,
				output_configuration_specific.feature_map_count,
				&alpha,
				*data[0],
				input_configuration_specific.feature_map_count,
				*additional_buffers[1],
				output_configuration_specific.feature_map_count,
				&beta,
				*additional_buffers[0],
				input_configuration_specific.feature_map_count));

			cuda_util::transpose(
				*cuda_config,
				*additional_buffers[0],
				*input_errors_buffer,
				input_configuration_specific.feature_map_count,
				input_elem_count_per_feature_map,
				entry_count,
				stream_id);
		}

		void convolution_1x1_layer_updater_cuda::enqueue_update_weights(
			unsigned int offset_input_entry_id,
			hipStream_t stream_id,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& gradient,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data_custom,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			cuda_linear_buffer_device_smart_ptr output_errors_buffer,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			std::vector<cuda_memobject_smart_ptr>& dynamic_memobjects,
			unsigned int entry_count,
			bool force_deterministic)
		{
			// Update weights
			{
				cuda_util::transpose(
					*cuda_config,
					*output_errors_buffer,
					*additional_buffers[1],
					output_elem_count_per_feature_map,
					output_configuration_specific.feature_map_count,
					entry_count,
					stream_id);

				cublas_safe_call(hipblasSetStream(cuda_config->get_cublas_handle(), stream_id));
				float alpha = 1.0F;
				float beta = 1.0F;
				cublas_safe_call(hipblasSgemm(
					cuda_config->get_cublas_handle(),
					HIPBLAS_OP_N,
					HIPBLAS_OP_T,
					input_configuration_specific.feature_map_count,
					output_configuration_specific.feature_map_count,
					entry_count * input_elem_count_per_feature_map,
					&alpha,
					*additional_buffers[0],
					input_configuration_specific.feature_map_count,
					*additional_buffers[1],
					output_configuration_specific.feature_map_count,
					&beta,
					*gradient[0],
					input_configuration_specific.feature_map_count));
			}

			// Update bias
			{
				cudnn_safe_call(hipdnnSetStream(cuda_config->get_cudnn_handle(), stream_id));
				cudnn_safe_call(hipdnnSetTensor4dDescriptor(
					output_data_desc,
					HIPDNN_TENSOR_NCHW,
					HIPDNN_DATA_FLOAT,
					entry_count,
					output_configuration_specific.feature_map_count,
					1,
					output_elem_count_per_feature_map));

				float alpha = 1.0F;
				float beta = 1.0F;
				cudnn_safe_call(hipdnnConvolutionBackwardBias(
					cuda_config->get_cudnn_handle(),
					&alpha,
					output_data_desc,
					*output_errors_buffer,
					&beta,
					bias_desc,
					*gradient[1]));
			}
		}

		void convolution_1x1_layer_updater_cuda::updater_configured()
		{
			cudnn_safe_call(hipdnnSetTensor4dDescriptor(
				bias_desc,
				HIPDNN_TENSOR_NCHW,
				HIPDNN_DATA_FLOAT,
				1,
				output_configuration_specific.feature_map_count,
				1,
				1));
		}

		std::vector<size_t> convolution_1x1_layer_updater_cuda::get_sizes_of_additional_buffers_per_entry() const
		{
			std::vector<size_t> res;

			res.push_back(input_elem_count_per_entry * sizeof(float));
			res.push_back(output_elem_count_per_entry * sizeof(float));

			return res;
		}

		bool convolution_1x1_layer_updater_cuda::is_in_place_backprop() const
		{
			return false;
		}
	}
}
