#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2015 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "hyperbolic_tangent_layer_updater_cuda.h"

#include <hip/hip_runtime.h>

#include "../hyperbolic_tangent_layer.h"
#include "../neural_network_exception.h"
#include "../nn_types.h"

#include "util_cuda.h"

static __forceinline__ __device__ float hyperbolic_tangent(
	float x,
	float hyperbolic_tangent_steepness2,
	float hyperbolic_tangent_major_multiplier)
{
	float y = __expf(x * hyperbolic_tangent_steepness2);
	return __fdividef(y - 1.0F, y + 1.0F) * hyperbolic_tangent_major_multiplier;
}

__global__ void hyperbolic_tangent_upd_kernel(
	const float4 * __restrict input,
	float4 * __restrict output,
	float hyperbolic_tangent_steepness2,
	float hyperbolic_tangent_major_multiplier,
	int elem_count)
{
	int elem_id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
	if (elem_id < elem_count)
	{
		float4 val = input[elem_id];
		val.x = hyperbolic_tangent(val.x, hyperbolic_tangent_steepness2, hyperbolic_tangent_major_multiplier);
		val.y = hyperbolic_tangent(val.y, hyperbolic_tangent_steepness2, hyperbolic_tangent_major_multiplier);
		val.z = hyperbolic_tangent(val.z, hyperbolic_tangent_steepness2, hyperbolic_tangent_major_multiplier);
		val.w = hyperbolic_tangent(val.w, hyperbolic_tangent_steepness2, hyperbolic_tangent_major_multiplier);
		output[elem_id] = val;
	}
}

static __forceinline__ __device__ float hyperbolic_tangent_deriviative(
	float x,
	float hyperbolic_tangent_major_multiplier_reverted,
	float hyperbolic_tangent_steepness3)
{
	float normalized_value = x * hyperbolic_tangent_major_multiplier_reverted;
	return hyperbolic_tangent_steepness3 * (1.0F - (normalized_value * normalized_value));
}

__global__ void hyperbolic_tangent_deriviative_upd_kernel(
	float4 * __restrict input_errors,
	const float4 * __restrict output_errors,
	const float4 * __restrict output_neurons,
	float hyperbolic_tangent_major_multiplier_reverted,
	float hyperbolic_tangent_steepness3,
	bool add_update_to_destination,
	int elem_count)
{
	int elem_id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
	if (elem_id < elem_count)
	{
		float4 val = output_neurons[elem_id];
		val.x = hyperbolic_tangent_deriviative(val.x, hyperbolic_tangent_major_multiplier_reverted, hyperbolic_tangent_steepness3);
		val.y = hyperbolic_tangent_deriviative(val.y, hyperbolic_tangent_major_multiplier_reverted, hyperbolic_tangent_steepness3);
		val.z = hyperbolic_tangent_deriviative(val.z, hyperbolic_tangent_major_multiplier_reverted, hyperbolic_tangent_steepness3);
		val.w = hyperbolic_tangent_deriviative(val.w, hyperbolic_tangent_major_multiplier_reverted, hyperbolic_tangent_steepness3);
		float4 current_error = output_errors[elem_id];
		float4 current_dst;
		if (add_update_to_destination)
		{
			current_dst = input_errors[elem_id];
			current_dst.x += current_error.x * val.x;
			current_dst.y += current_error.y * val.y;
			current_dst.z += current_error.z * val.z;
			current_dst.w += current_error.w * val.w;
		}
		else
		{
			current_dst.x = current_error.x * val.x;
			current_dst.y = current_error.y * val.y;
			current_dst.z = current_error.z * val.z;
			current_dst.w = current_error.w * val.w;
		}
		input_errors[elem_id] = current_dst;
	}
}

namespace nnforge
{
	namespace cuda
	{
		hyperbolic_tangent_layer_updater_cuda::hyperbolic_tangent_layer_updater_cuda()
		{
		}

		hyperbolic_tangent_layer_updater_cuda::~hyperbolic_tangent_layer_updater_cuda()
		{
		}

		void hyperbolic_tangent_layer_updater_cuda::enqueue_forward_propagation(
			hipStream_t stream_id,
			cuda_linear_buffer_device::ptr output_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device::ptr>& data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data_custom,
			const std::vector<cuda_linear_buffer_device::const_ptr>& input_buffers,
			const std::vector<cuda_linear_buffer_device::const_ptr>& persistent_working_data,
			cuda_linear_buffer_device::ptr temporary_working_fixed_buffer,
			cuda_linear_buffer_device::ptr temporary_working_per_entry_buffer,
			cuda_linear_buffer_device::ptr temporary_per_entry_buffer,
			unsigned int entry_count)
		{
			int elem_count = (output_elem_count_per_entry * entry_count + 3) / 4;
			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				elem_count);
			hyperbolic_tangent_upd_kernel<<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
				*input_buffers[0],
				*output_buffer,
				hyperbolic_tangent_steepness2,
				hyperbolic_tangent_major_multiplier,
				elem_count);
		}

		void hyperbolic_tangent_layer_updater_cuda::enqueue_backward_data_propagation(
			hipStream_t stream_id,
			unsigned int input_index,
			cuda_linear_buffer_device::ptr input_errors_buffer,
			cuda_linear_buffer_device::const_ptr output_errors_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device::ptr>& data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data_custom,
			const std::vector<cuda_linear_buffer_device::const_ptr>& input_neurons_buffers,
			cuda_linear_buffer_device::const_ptr output_neurons_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& persistent_working_data,
			cuda_linear_buffer_device::ptr temporary_working_fixed_buffer,
			cuda_linear_buffer_device::ptr temporary_working_per_entry_buffer,
			cuda_linear_buffer_device::const_ptr temporary_per_entry_buffer,
			bool add_update_to_destination,
			unsigned int entry_count)
		{
			int elem_count = (output_elem_count_per_entry * entry_count + 3) / 4;
			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				elem_count);
			hyperbolic_tangent_deriviative_upd_kernel<<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
				*input_errors_buffer,
				*output_errors_buffer,
				*output_neurons_buffer,
				hyperbolic_tangent_major_multiplier_reverted,
				hyperbolic_tangent_steepness3,
				add_update_to_destination,
				elem_count);
		}

		void hyperbolic_tangent_layer_updater_cuda::updater_configured()
		{
			nnforge_shared_ptr<const hyperbolic_tangent_layer> layer_derived = nnforge_dynamic_pointer_cast<const hyperbolic_tangent_layer>(layer_schema);

			hyperbolic_tangent_steepness2 = layer_derived->steepness * 2.0F;
			hyperbolic_tangent_major_multiplier = layer_derived->scale;
			hyperbolic_tangent_steepness3 = layer_derived->steepness * layer_derived->scale;
			hyperbolic_tangent_major_multiplier_reverted = 1.0F / layer_derived->scale;
		}

		int hyperbolic_tangent_layer_updater_cuda::get_input_index_layer_can_write(const layer_action& action) const
		{
			return 0;
		}

		bool hyperbolic_tangent_layer_updater_cuda::is_backward_data_dependent_on_input_buffer(unsigned int action_input_index, unsigned int data_input_index) const
		{
			return false;
		}

		bool hyperbolic_tangent_layer_updater_cuda::is_backward_data_dependent_on_output_buffer(unsigned int action_input_index) const
		{
			return true;
		}
	}
}
